#include "hip/hip_runtime.h"
/*******************************************************************************

This program uses the Thrust library to perform vector arithmetic .

Author: Said Darham
*******************************************************************************/

#include <iostream>
#include <stdlib.h> //srand and rand
#include <math.h>

//Thrust libraries headers
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>


//Timer struct declaration. Using CUDA EVENTS
typedef struct timer{
  hipEvent_t startEvent;
  hipEvent_t stopEvent;
  float time_ms;
} timerEvent;

/*******************************************************************************

                        PROFILER FUNCTIONS USING EVENTS

*******************************************************************************/
void startEventTimer(timerEvent *timer){
  /* startEventTimer()
     Creates and starts recording an event
  */
  hipEventCreate(&timer->startEvent);
  hipEventCreate(&timer->stopEvent);
  hipEventRecord(timer->startEvent);
}

void stopEventTimer(timerEvent *timer){
  /* stopEventTimer()
     Stops an event and calculates the elapsed time between start and stop event
  */
  hipEventRecord(timer->stopEvent);
  hipEventSynchronize(timer->stopEvent);
  hipEventElapsedTime(&timer->time_ms, timer->startEvent, timer->stopEvent);

}
void freeEventTimer(timerEvent *timer){
  /*  freeEventTimer()
      cleans up the events
  */
  hipEventDestroy(timer->startEvent);
  hipEventDestroy(timer->stopEvent);
}

void checkDevices(void){
  //Check and print devices name
  hipDeviceProp_t prop;
  int deviceCount; //number of devices found
  int devId = 0; // default device Id

  hipGetDeviceCount(&deviceCount);

  if(deviceCount == 0){
    std::cout << "No GPU Device Found\n";
    exit(0);
  }else if(deviceCount == 1){
    hipSetDevice(devId); //set the device 0 as default
  }

  std::cout << "Number Of Devices Found: " << deviceCount << std::endl;
  //Print device names and some basic associated properties
  for (int i = 0; i<deviceCount; i++){
    hipGetDeviceProperties(&prop,i);
    std::cout << "Device " << i << " Name: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
  }
}

void printArray(thrust::host_vector<int> array, int n){
    //helper function to Print the array of n elements and what function is used
    for(int i = 0; i<10; i++){
      std::cout << array[i] << ' ';
    }
    std::cout << std::endl;
}


/*******************************************************************************

ARITHMETIC KERNEL FUNCTIONS

*******************************************************************************/
// Add Function
__global__ void add(int *a, int *b, int *c, int n){
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
  c[id] = a[id] + b[id];
}
// subtract function
__global__ void subtract(int *a, int *b, int *c, int n){
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
  c[id] = a[id] - b[id];
}
// multiply function
__global__ void mult(int *a, int *b, int *c, int n){
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
  c[id] = a[id] * b[id];
}
// Moudulu function
__global__ void mod(int *a, int *b, int *c, int n){
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
  c[id] = a[id] % b[id];
}

__host__ static __inline__ int myRand(){
  //to be used with thrust's generate() to supply with random numbers b/w 0-3
  return ((int)rand() % 4);
}



/*******************************************************************************

CUDA KERNELS TEST

*******************************************************************************/
void executeCudaTest(int numBlocks, int blockSize, int totalThreads){

  std::cout << "\n\t\t*****Executing Arithmetic Functions Using CUDA kernels*****" << std::endl;

  // Host input/output vectors
  int *h_a, *h_b, *h_c_add,*h_c_sub,*h_c_mult,*h_c_mod;

  // Device input/output vectors
  int *d_a, *d_b, *d_c_add,*d_c_sub,*d_c_mult,*d_c_mod;

  // Size, in bytes, of each vector
  const unsigned int bytes = totalThreads*sizeof(int);

  // Allocate memory for each vector on host Pinned
  hipHostMalloc((void**)&h_a, bytes);
  hipHostMalloc((void**)&h_b, bytes);
  hipHostMalloc((void**)&h_c_add, bytes);
  hipHostMalloc((void**)&h_c_sub, bytes);
  hipHostMalloc((void**)&h_c_mult, bytes);
  hipHostMalloc((void**)&h_c_mod, bytes);

  // Allocate memory for each vector on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c_add, bytes);
  hipMalloc(&d_c_sub, bytes);
  hipMalloc(&d_c_mult, bytes);
  hipMalloc(&d_c_mod, bytes);

  //initialize the input vectors
	for(int i = 0;i<totalThreads;i++){
		//first array is 0 through number of threads
		h_a[i] = i;
		// second array is a random number between 0 and 3
		h_b[i] = rand() % 4;
	}

  //create a struct which will contain info for timing using events
  timerEvent timer;

  //Transfer and Profile data from host to device and profile using EVENTS
  startEventTimer(&timer);
  hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

  //Execute  the kernel arithmetic functions
  add<<<numBlocks, blockSize>>>(d_a, d_b, d_c_add, totalThreads);
  subtract<<<numBlocks, blockSize>>>(d_a, d_b, d_c_sub, totalThreads);
  mult<<<numBlocks, blockSize>>>(d_a, d_b, d_c_mult, totalThreads);
  mod<<<numBlocks, blockSize>>>(d_a, d_b, d_c_mod, totalThreads);

  //Transfer data from device to host
  hipMemcpy(h_c_add, d_c_add, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_c_sub, d_c_sub, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_c_mult, d_c_mult, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_c_mod, d_c_mod, bytes, hipMemcpyDeviceToHost);
  stopEventTimer(&timer);

  std::cout << "Time Elaplsed For CUDA kernels: " << timer.time_ms << " ms" << std::endl;

  //destroy Event timer
  freeEventTimer(&timer);

  //free up space on our GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c_add);
	hipFree(d_c_sub);
	hipFree(d_c_mult);
	hipFree(d_c_mod);

	//free up space on our CPU use hipHostFree since pinnned
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c_add);
	hipHostFree(h_c_sub);
	hipHostFree(h_c_mult);
	hipHostFree(h_c_mod);
}


/*******************************************************************************

THRUST TEST

*******************************************************************************/
void executeThrustTest(int totalThreads){

  std::cout << "\n\t\t*****Executing Arithmetic Functions Using Thrust*****" << std::endl;

  // host vectors
  thrust::host_vector<int> h_a(totalThreads);
  thrust::host_vector<int> h_b(totalThreads);

  // device vectors
  thrust::device_vector<int> d_c_add(totalThreads);
  thrust::device_vector<int> d_c_sub(totalThreads);
  thrust::device_vector<int> d_c_mult(totalThreads);
  thrust::device_vector<int> d_c_mod(totalThreads);

  //Initialize data arrays
  for( int i = 0; i < totalThreads; i++)
    h_a[i] = i;

  //generate random data on the host_vector
  thrust::generate(h_b.begin(), h_b.end(), myRand);

  //create a struct which will contain info for timing using events
  timerEvent timer;
  startEventTimer(&timer);

  //copy vectors from host to devices
  thrust::device_vector<int> d_a = h_a;
  thrust::device_vector<int> d_b = h_b;

  //perform arithmetic functions
  thrust::transform(d_a.begin(), d_a.end(), d_b.begin(), d_c_add.begin(), thrust::plus<int>());
  thrust::transform(d_a.begin(), d_a.end(), d_b.begin(), d_c_sub.begin(), thrust::minus<int>());
  thrust::transform(d_a.begin(), d_a.end(), d_b.begin(), d_c_mult.begin(), thrust::multiplies<int>());
  thrust::transform(d_a.begin(), d_a.end(), d_b.begin(), d_c_mod.begin(), thrust::modulus<int>());

  //copy results from device to host
  thrust::host_vector<int> h_c_add = d_c_add;
  thrust::host_vector<int> h_c_sub = d_c_sub;
  thrust::host_vector<int> h_c_mult = d_c_mult;
  thrust::host_vector<int> h_c_mod = d_c_mod;

  stopEventTimer(&timer);

  std::cout << "Time Elaplsed For Arithmetic using Thrust: " << timer.time_ms << " ms" << std::endl;

  //destroy Event timer
  freeEventTimer(&timer);
}



/*******************************************************************************

MAIN

*******************************************************************************/
int main(int argc, char** argv)
{

  int totalThreads = (1 << 10);
  int blockSize = 256;

  //User wants to run the Global vs Pinned Examples
  if( argc > 2 && argc < 4){
    // Ensure the user supplies both number of threads and block size
    // otherwise use default values
    totalThreads = atoi(argv[1]);
    blockSize = atoi(argv[2]);
  }

  int numBlocks = totalThreads/blockSize;
  std::cout << "\nUsing " << totalThreads << " Threads and " << blockSize << " BlockSize\n" ;

  // validate command line arguments
  if (totalThreads % blockSize != 0) {
    ++numBlocks;
    totalThreads = numBlocks*blockSize;
    std::cout << "Warning: Total thread count is not evenly divisible by the block size\n";
    std::cout << "The total number of threads will be rounded up to %d\n";
  }

  // get number of devices and print some basic properties
  checkDevices();

  executeCudaTest( numBlocks, blockSize, totalThreads);

  executeThrustTest( totalThreads );

  return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void what_is_my_id_2d_A(
				unsigned int * const block_x,
				unsigned int * const block_y,
				unsigned int * const thread,
				unsigned int * const calc_thread,
				unsigned int * const x_thread,
				unsigned int * const y_thread,
				unsigned int * const grid_dimx,
				unsigned int * const block_dimx,
				unsigned int * const grid_dimy,
				unsigned int * const block_dimy)
{
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

	block_x[thread_idx] = blockIdx.x;
	block_y[thread_idx] = blockIdx.y;
	thread[thread_idx] = threadIdx.x;
	calc_thread[thread_idx] = thread_idx;
	x_thread[thread_idx] = idx;
	y_thread[thread_idx] = idy;
	grid_dimx[thread_idx] = gridDim.x;
	block_dimx[thread_idx] = blockDim.x;
	grid_dimy[thread_idx] = gridDim.y;
	block_dimy[thread_idx] = blockDim.y;
}

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16

#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))

/* Declare statically six arrays of ARRAY_SIZE each */
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

int main(void)
{
	/* Total thread count = 32 * 4 = 128 */
	const dim3 threads_rect(32,4);
	const dim3 blocks_rect(1,4);

	/* Total thread count = 16 * 8 = 128 */
	const dim3 threads_square(16, 8); /* 16 * 8 */
	const dim3 blocks_square(2,2);

	/* Needed to wait for a character at exit */
	char ch;

	/* Declare statically six arrays of ARRAY_SIZE each */
	unsigned int * gpu_block_x;
	unsigned int * gpu_block_y;
	unsigned int * gpu_thread;
	unsigned int * gpu_warp;
	unsigned int * gpu_calc_thread;
	unsigned int * gpu_xthread;
	unsigned int * gpu_ythread;
	unsigned int * gpu_grid_dimx;
	unsigned int * gpu_block_dimx;
	unsigned int * gpu_grid_dimy;
	unsigned int * gpu_block_dimy;

	/* Allocate arrays on the GPU */
	hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

	for( int kernel= 0; kernel < 2; kernel++)
	{
		switch(kernel)
		{
			case 0:
			{
				/* Execute our kernel */
				what_is_my_id_2d_A<<<blocks_rect, threads_rect>>>(gpu_block_x, gpu_block_y,
	gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
	gpu_grid_dimy, gpu_block_dimy);
			} break;

			case 1:
			{
				/* Execute our kernel */
				what_is_my_id_2d_A<<<blocks_square, threads_square>>>(gpu_block_x, gpu_block_y,
	gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
	gpu_grid_dimy, gpu_block_dimy);
			} break;

			default: exit(1); break;
		}

		/* Copy back the gpu results to the CPU */
		hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

		printf("\nKernel %d\n", kernel);
		/* Iterate through the arrays and print */
		for(int y = 0; y < ARRAY_SIZE_Y; y++)
		{
			for(int x = 0; x < ARRAY_SIZE_X; x++)
			{
				printf("CT: %2u BKX: %1u BKY: %1u TID: %2u YTID: %2u XTID: %2u GDX: %1u BDX: %1u GDY: %1u BDY: %1u\n",
						cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x], cpu_thread[y][x], cpu_ythread[y][x],
						cpu_xthread[y][x], cpu_grid_dimx[y][x], cpu_block_dimx[y][x], cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);

			}
		}


		/* Free the arrays on the GPU as now we're done with them */
		hipFree(gpu_block_x);
		hipFree(gpu_block_y);
		hipFree(gpu_thread);
		hipFree(gpu_warp);
		hipFree(gpu_calc_thread);
		hipFree(gpu_xthread);
		hipFree(gpu_ythread);
		hipFree(gpu_grid_dimy);
		hipFree(gpu_block_dimy);
	}
}

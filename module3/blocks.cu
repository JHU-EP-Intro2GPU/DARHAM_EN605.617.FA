
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_SIZE 256
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

/* Declare  statically two arrays of ARRAY_SIZE each */
unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];


__global__
void what_is_my_id(unsigned int * block, unsigned int * thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

void main_sub0()
{

	/* Declare pointers for GPU based params */
	unsigned int *gpu_block;
	unsigned int *gpu_thread;

	hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMemcpy( cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );
	hipMemcpy( cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );

	const unsigned int num_blocks = ARRAY_SIZE/16;
	const unsigned int num_threads = ARRAY_SIZE/num_blocks;

	/* Execute our kernel */
	what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread);

	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy( cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost );
	hipMemcpy( cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost );
	hipFree(gpu_block);
	hipFree(gpu_thread);

	/* Iterate through the arrays and print */
	for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		printf("Thread: %2u - Block: %2u\n",cpu_thread[i],cpu_block[i]);
	}
}

int main()
{
	main_sub0();

	return EXIT_SUCCESS;
}

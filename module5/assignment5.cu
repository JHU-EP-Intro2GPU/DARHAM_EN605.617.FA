//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> //srand and rand
#include <math.h>

// Constant data declaration
#define WORKSIZE 1024 // define a default worksize for constant data
__device__ __constant__ int d_a_const[WORKSIZE];
__device__ __constant__ int d_b_const[WORKSIZE];



/*
	Profile functions. Taken and modified from https://devblogs.nvidia.com/how-optimize-data-transfers-cuda-cc/
*/
void profileCopiesHostToDevice(int        *d_a,
                               int        *h_a,
                               int        *d_b,
				                       int        *h_b,
                               const unsigned int  bytes,
                               const char         *desc){

  // events for timing
  hipEvent_t startEvent, stopEvent;

  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

  //start a recording event and execute the transfer afte
  hipEventRecord(startEvent, 0);
  // Use either cudaMemcpy or cudaMemcpyToSymbol depending on shared vs constant memory
  if(strcmp(desc, "Shared") == 0){
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
  }else if(strcmp(desc, "Constant") == 0){
    hipMemcpyToSymbol( HIP_SYMBOL(d_a_const), h_a, bytes,0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol( HIP_SYMBOL(d_b_const), h_b, bytes,0, hipMemcpyHostToDevice);
  }

  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);

  float time;
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf("\nTransfers Host to Device Time Elaped: %f ms, Bandwidth (MB/s): %f\n\n", time, bytes * 1e-3 / time);

  // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}

void profileCopiesDeviceToHost( int *h_c_add, int *d_c_add, int *h_c_sub, int *d_c_sub,
								                int *h_c_mult, int *d_c_mult, int *h_c_mod, int *d_c_mod,
                                const unsigned int bytes, const char *desc){


  // events for timing
  hipEvent_t startEvent, stopEvent;

  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

  hipEventRecord(startEvent, 0);


  if(strcmp(desc,"Shared") == 0){
    hipMemcpy( h_c_add, d_c_add, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_sub, d_c_sub, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_mult, d_c_mult, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_mod, d_c_mod, bytes, hipMemcpyDeviceToHost);
  }else if(strcmp(desc,"Constant") == 0){
    hipMemcpy( h_c_add, d_c_add, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_sub, d_c_sub, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_mult, d_c_mult, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_c_mod, d_c_mod, bytes, hipMemcpyDeviceToHost);
  }

  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);

  float time;
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf("\n%s transfers Device To Host Time Elaped: %f ms, Bandwidth (MB/s): %f\n\n",desc,time, bytes * 1e-3 / time);

  // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}


/*
				Arithmetic Functions Using shared Memory
*/
// Add Function
__global__ void add_shared(int *a, int *b, int *c, int n){
    extern __shared__ int res[];
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n)
        res[threadIdx.x] = a[id] + b[id];

	__syncthreads(); // wait for all threads in the block to finish
	c[threadIdx.x] = res[threadIdx.x];//since threads from different blocks cannot talk, use thread index instead
}

// subtract function
__global__ void subtract_shared(int *a, int *b, int *c, int n){
   extern __shared__ int res[];

    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        res[threadIdx.x] = a[id] - b[id];

	__syncthreads(); // wait for all threads in the block to finish

	c[threadIdx.x] = res[threadIdx.x];

}

// multiply function
 __global__ void mult_shared(int *a, int *b, int *c, int n){
    extern __shared__ int res[];
     // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        res[threadIdx.x] = a[id] * b[id];

	__syncthreads(); // wait for all threads in the block to finish

	c[threadIdx.x] = res[threadIdx.x];

}

// Moudulus function
__global__ void mod_shared(int *a, int *b, int *c, int n){
 extern __shared__ int res[];

    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        res[threadIdx.x] = a[id] % b[id];

	__syncthreads(); // wait for all threads in the block to finish

	c[threadIdx.x] = res[threadIdx.x];
}



/*
				Arithmetic Functions Using Constant Memory
*/
// Add Function
__global__ void add_const( int *c, int n){
    // Get our global thread ID
    const unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    // use the constant data declared
    if (id < n)
        c[id] = d_a_const[id] + d_b_const[id];
}

// subtract function
__global__ void subtract_const(int *c, int n){
    // Get our global thread ID
    const unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = d_a_const[id] - d_b_const[id];
}

// multiply function
 __global__ void mult_const(int *c, int n){
    // Get our global thread ID
    const unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = d_a_const[id] * d_b_const[id];
}

// Moudulus function
__global__ void mod_const(int *c, int n){

    // Get our global thread ID
    const unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = d_a_const[id] % d_b_const[id];
}



/*
  Function calls to arithmetic functions using shared memory and timing
*/
void perform_add_shared(int numBlocks, int totalThreads, int *d_a, int *d_b, int *d_c_add){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing add function
  printf("  Performing Add function...");
  add_shared<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_a, d_b, d_c_add, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}

void perform_sub_shared(int numBlocks, int totalThreads, int *d_a,int *d_b,int *d_c_sub){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing subtract function
  printf("  Performing subtract function");
  subtract_shared<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_a, d_b, d_c_sub, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

}

void perform_mult_shared(int numBlocks, int totalThreads, int *d_a,int *d_b,int *d_c_mult){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing mult function
  printf("  Performing mult function");
  mult_shared<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_a, d_b, d_c_mult, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

}

void perform_mod_shared(int numBlocks, int totalThreads, int *d_a,int *d_b,int *d_c_mod){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing mod fuction
  printf("  Performing mod function");
  mod_shared<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_a, d_b, d_c_mod, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}


// Shared Memory Implementation function
void execute_arithmetic_sharedMem(int totalThreads, int numBlocks){

printf("\t\t*****Executing Arithmetic Functions Using Shared Memory*****\n");

	// Host input vectors
	int *h_a, *h_b;
	//Host output vectors for different functions "h_c_func"
	int *h_c_add,*h_c_sub,*h_c_mult,*h_c_mod;

	// Device input vectors
	int *d_a, *d_b;
	//Device output vector
	int *d_c_add,*d_c_sub,*d_c_mult,*d_c_mod;

	// Size, in bytes, of each vector
	const unsigned int bytes = totalThreads*sizeof(int);

	// Allocate memory for each vector on host Pinned
	hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_b, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_add, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_sub, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_mult, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_mod, bytes, hipHostMallocDefault);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c_add, bytes);
	hipMalloc(&d_c_sub, bytes);
	hipMalloc(&d_c_mult, bytes);
	hipMalloc(&d_c_mod, bytes);

	//initialize the input vectors
	for(int i = 0;i<totalThreads;i++){
		//first array is 0 through number of threads
		h_a[i] = i;
		// second array is a random number between 0 and 3
		h_b[i] = rand() % 4;
	}

  //printf the first 7 elements of input arrays
  printf("Array 1: ");
  for(int i = 0; i<7; i++){
   printf("%d ", h_a[i]);
  }
  printf("\nArray 2: ");
  for(int i = 0; i<7; i++){
    printf("%d ", h_b[i]);
  }
  printf("\n\n");

	//copy both input arrays from host to device and profile it (see profileCopiesHostToDevice)
	profileCopiesHostToDevice(d_a, h_a, d_b, h_b, bytes, "Shared");

  //Perform arithmetic functions
  perform_add_shared(numBlocks, totalThreads, d_a, d_b, d_c_add);
  perform_sub_shared(numBlocks, totalThreads, d_a, d_b, d_c_sub);
  perform_mult_shared(numBlocks, totalThreads, d_a, d_b, d_c_mult);
  perform_mod_shared(numBlocks, totalThreads, d_a, d_b, d_c_mod);

	//copy the output arrays from device to host
	profileCopiesDeviceToHost(h_c_add,d_c_add,h_c_sub, d_c_sub,h_c_mult, d_c_mult,h_c_mod, d_c_mod, bytes,"Shared");

  // printf the first 7 elements of the results
  printf("Arithmetic Results: \n");
	printf("Add: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_add[i]);
	}
  printf("\nSubtract: ");

	for(int i = 0; i<7; i++){
		printf("%d ", h_c_sub[i]);
	}
  printf("\nMultiply: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_mult[i]);
	}
  printf("\nMultiply: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_mod[i]);
	}
	printf("\n\n");

	//free up space on our GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c_add);
	hipFree(d_c_sub);
	hipFree(d_c_mult);
	hipFree(d_c_add);

	//free up space on our CPU use cudaFreeHost since pinnned
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c_add);
 	hipHostFree(h_c_sub);
	hipHostFree(h_c_mult);
	hipHostFree(h_c_mod);
}

/*
  Function calls to arithmetic functions using constant memory
*/
void perform_add_const(int numBlocks, int totalThreads,int *d_c_add){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing add function
  printf("  Performing Add function...");
  add_const<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_c_add, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}

void perform_sub_const(int numBlocks, int totalThreads, int *d_c_sub){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing subtract function
  printf("  Performing subtract function");
  subtract_const<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_c_sub, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

}

void perform_mult_const(int numBlocks, int totalThreads,int *d_c_mult){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing mult function
  printf("  Performing mult function");
  mult_const<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>(d_c_mult, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

}

void perform_mod_const(int numBlocks, int totalThreads, int *d_c_mod){

  float time;
  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

	//start a recording event and execute the Kernels after
  hipEventRecord(startEvent, 0);
  //performing mod fuction
  printf("  Performing mod function");
  mod_const<<<numBlocks, totalThreads, totalThreads*sizeof(int)>>>( d_c_mod, totalThreads);
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0); //stop
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  printf(" Elapsed Time: %f\n", time);
   // clean up events
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}

// Constant Memory Implementation
void execute_arithmetic_constMem(int totalThreads, int numBlocks){


	printf("\t\t*****Executing Arithmetic Functions Using Constant Memory*****\n");
		// Host input vectors
	int *h_a, *h_b;
	//Host output vectors for different functions "h_c_func"
	int *h_c_add,*h_c_sub,*h_c_mult,*h_c_mod;

	// Device input vectors
	int *d_a, *d_b;
	//Device output vector
	int *d_c_add,*d_c_sub,*d_c_mult,*d_c_mod;

	// Size, in bytes, of each vector
	const unsigned int bytes = totalThreads*sizeof(int);

	// Allocate memory for each vector on host Pinned
	hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_b, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_add, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_sub, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_mult, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c_mod, bytes, hipHostMallocDefault);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c_add, bytes);
	hipMalloc(&d_c_sub, bytes);
	hipMalloc(&d_c_mult, bytes);
	hipMalloc(&d_c_mod, bytes);

	//initialize the input vectors
	for(int i = 0;i<totalThreads;i++){
		//first array is 0 through number of threads
		h_a[i] = i;
		// second array is a random number between 0 and 3
		h_b[i] = rand() % 4;
	}


  //printf the first 7 elements of input arrays
  printf("Array 1: ");
  for(int i = 0; i<7; i++){
   printf("%d ", h_a[i]);
  }
  printf("\nArray 2: ");
  for(int i = 0; i<7; i++){
    printf("%d ", h_b[i]);
  }
  printf("\n\n");


	//copy both input arrays from host to device using cudaMemcpyToSymbol() (see profileCopiesHostToDevice)
	profileCopiesHostToDevice(d_a_const, h_a, d_b_const, h_b, bytes, "Constant");

  //Perform arithmetic functions
  perform_add_const(numBlocks, totalThreads, d_c_add);
  perform_sub_const(numBlocks, totalThreads, d_c_sub);
  perform_mult_const(numBlocks, totalThreads, d_c_mult);
  perform_mod_const(numBlocks, totalThreads, d_c_mod);

	//copy the output arrays from device to host using cudaMemcyFromSymbol()
	profileCopiesDeviceToHost(h_c_add,d_c_add,h_c_sub, d_c_sub,h_c_mult, d_c_mult,h_c_mod, d_c_mod, bytes,"Constant");

  // printf the first 7 elements of the results
  printf("Arithmetic Results: \n");
	printf("Add: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_add[i]);
	}
  printf("\nSubtract: ");

	for(int i = 0; i<7; i++){
		printf("%d ", h_c_sub[i]);
	}
  printf("\nMultiply: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_mult[i]);
	}
  printf("\nMod: ");
	for(int i = 0; i<7; i++){
		printf("%d ", h_c_mod[i]);
	}
	printf("\n\n");

	//free up space on our GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c_add);
	hipFree(d_c_sub);
	hipFree(d_c_mult);
	hipFree(d_c_add);

	//free up space on our CPU use cudaFreeHost since pinnned
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c_add);
	hipHostFree(h_c_sub);
	hipHostFree(h_c_mult);
	hipHostFree(h_c_mod);

}


int main(int argc, char** argv)
{

	int totalThreads = (1 << 10);
	int blockSize = 256;

      //User wants to run the Global vs Pinned Examples
      if( argc > 2 && argc < 4){
        // Ensure the user supplies both number of threads and block size
        // otherwise use default values
        totalThreads = atoi(argv[1]);
        blockSize = atoi(argv[2]);
      }

    	int numBlocks = totalThreads/blockSize;

      printf("\nUsing %d Threads and %d BlockSize\n",totalThreads, blockSize);

    	// validate command line arguments
    	if (totalThreads % blockSize != 0) {
    		++numBlocks;
    		totalThreads = numBlocks*blockSize;

    		printf("Warning: Total thread count is not evenly divisible by the block size\n");
    		printf("The total number of threads will be rounded up to %d\n", totalThreads);
    	}

    	// Lets see what we are working with and calculate the Amount of data we are transfering
    	hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop,0);
    	const unsigned int bytes = totalThreads*sizeof(int);
    	printf("\nDevice: %s\n", prop.name);
    	printf("Transfer size (MB): %d\n\n", bytes * bytes / totalThreads);


    	//Execute Pageable Arithmetic
    	execute_arithmetic_sharedMem(totalThreads, numBlocks);
      //Execute The Pinned Arithmetic
      execute_arithmetic_constMem(totalThreads, numBlocks);



  return 0;
}
